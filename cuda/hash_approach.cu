#include "hip/hip_runtime.h"
#include "general.cuh"
#include <stdio.h>
#include<iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <stdio.h>

__global__ void partitionKernelHash(int * c1, int * c2,int n1, int n2, int partitions, int blocks, CustomContainer * containers1, CustomContainer * containers2){
    int blockSize1 = n1 / blocks;
    int p;
    int idx = blockSize1 * blockIdx.x + threadIdx.x;
    for (int i = idx; i < blockSize1 * (blockIdx.x + 1) && i < n1; i += blockDim.x){
        p = c1[i] % partitions;
        containers1->push_back(p, 0, c1[i]);
    }
    int blockSize2 = n2 / blocks;
    idx = blockSize2 * blockIdx.x + threadIdx.x;
    for (int i = idx; i < blockSize2 * (blockIdx.x + 1) && i < n2; i += blockDim.x){
        p = c2[i] % partitions;
        containers2->push_back(p, 0, c2[i]);
    }
}
__global__ void probe(CustomContainer * container1, CustomContainer * container2, int partitions, int * finalCount){
    __shared__ int countArr[256];
    int * col1 = container1->getPartition(blockIdx.x);
    int * col2 = container2->getPartition(blockIdx.x);
    int n1 = container1->getPartitionSize(blockIdx.x);
    int n2 = container2->getPartitionSize(blockIdx.x);
    int idx = threadIdx.x;
    int count = 0;
    for(int i = idx; i < n1; i += blockDim.x){
        for (int j = 0; j < n2; j++){
            count += (col1[i] == col2[j]);
        }
    }
    countArr[threadIdx.x] = count;
    __syncthreads();
    if(threadIdx.x == 0){
        count = 0;
        for(int i = 0; i < blockDim.x; i++){
            count += countArr[i];
        }
        //printf("%d\n", count);
        atomicAdd(finalCount, count);
    }
}

void partitionedHash(int * c1, int * c2, int n1, int n2, int partitions, bool serial) { 
    thrust::device_vector<int> dCol1(c1, c1 + n1);
    thrust::device_vector<int> dCol2(c2, c2 + n2);
    int blocks = 80;
    int blockSize = n1/1;
    CustomContainer * containers1 = new CustomContainer(2 * n1/partitions, partitions, 1, NULL);
    CustomContainer * containers2 = new CustomContainer(2 * n2/partitions, partitions, 1, NULL);
    CustomContainer * deviceContainers1 = 0;
    CustomContainer * deviceContainers2 = 0;
    hipMalloc(&deviceContainers1, sizeof(CustomContainer));
    hipMalloc(&deviceContainers2, sizeof(CustomContainer));
    hipMemcpy(&deviceContainers1->container, &containers1->container, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers1->meta, &containers1->meta, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers1->head, &containers1->head, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers2->container, &containers2->container, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers2->meta, &containers2->meta, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers2->head, &containers2->head, sizeof(int *), hipMemcpyHostToDevice);
    int * finalCount = 0;
    hipMalloc(&finalCount, sizeof(int)*partitions);
    try{
        partitionKernelHash<<<blocks, 256>>>(thrust::raw_pointer_cast(dCol1.data()),thrust::raw_pointer_cast(dCol2.data()),n1,n2, partitions, blocks, deviceContainers1, deviceContainers2);
    } catch (thrust::system_error e){
        std::cout << "Error: " << e.what() << std::endl;
    }
    hipDeviceSynchronize();
    probe<<<partitions, 256>>>(deviceContainers1,deviceContainers2,partitions, finalCount); 
    hipDeviceSynchronize();
    int finalCountHost = 0;
    hipMemcpy(&finalCountHost, finalCount, sizeof(int), hipMemcpyDeviceToHost);
    printf("Final Count: %d\n", finalCountHost);
}