#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <vector>
#include <unordered_set>
#include <algorithm>
#include <string>
#include <unordered_map>
#include <string>
#include <functional>
#include <math.h>
#include <emmintrin.h>
#include <string.h>
#include<chrono>
#include <omp.h>
#include "general.cuh"
#include "hash_approach.cuh"
#include "sort_approach.cuh"
using namespace std;
int pNo = 127;

void loopOverBlock(int * arr1, int * arr2, int & c1, int & c2, int blkSize){
    int cnt = 0;
    int N = c1 / blkSize;
    for (int blkNo = 0; blkNo < N; blkNo++){
        for (int i = 0; i < c1; i++){
            for (int j = blkNo * blkSize; j < (blkNo + 1) * blkSize; j++){
                if (arr1[i] == arr2[j]){
                    cnt += 1;
                }
            }
        }
    }
    cout << cnt << endl;
}

void basicLoop(int * arr1, int * arr2, int & c1, int & c2){
    int cnt = 0;
        for (int i = 0; i < c1; i++){
            for (int j = 0; j < c2; j++){
                if (arr1[i] == arr2[j]){
                    cnt += 1;
                }
            }
        }
    cout << cnt << endl;
}

void readFile(char * file1, char * file2, int * col1, int * col2, int *c1, int *c2, int inputSize){
    char * file;
    int * container;
    ifstream reader1(file1);
    ifstream reader2(file2);
    for (int f = 0; f < 2; f++){
        int * c;
        ifstream * reader;
        if (f == 0){
            file = file1;
            container = col1;
            c = c1;
            reader = &reader1;
        } else {
            file = file2;
            container = col2;
            c = c2;
            reader = &reader2;
        }
        
        int lc = 0;
        int item = 0;
        char buf[2048];
        while(!reader->eof() && *c < inputSize)
        {
            reader->read(buf, sizeof(buf));
            uint64_t k = reader->gcount();
            for (uint64_t i = 0; i < k && *c < inputSize; ++i)
            {
                switch (buf[i])
                {
                    case '\r':
                        break;
                    case '\n':
                        item = 0;
                        break;
                    case ' ':
                        container[*c] = item;
                        *c = *c + 1;
                        item = 0;
                        break;
                    case '0': case '1': case '2': case '3':
                    case '4': case '5': case '6': case '7':
                    case '8': case '9':
                        item = 10*item + buf[i] - '0';
                        break;
                    default:
                        std::cerr << "Bad format\n";
                }    
            }
        }
            __builtin_prefetch(container + *c + 32, 1, 0);
            reader->close();
            reader->clear();
    }
}

int threads = 1;
int main(int argc, char ** argv){
    if (argc < 7){
        cout << "Wrong number of argument. Use ./join num_of_elements func_name fileA fileB operator";
        return 0;
    }
    int cacheSize = 12000;
    int partitionSize = 1200;
    int inputSize = pow(10, stoi(argv[1]));
    int * col1 = new int[inputSize + inputSize/100];
    int * col2 = new int[inputSize + inputSize/100];
    int * dCol1 = 0;
    int * dCol2 = 0;
    char * funcName = argv[2];
    char * file1 = argv[3];
    char * file2 = argv[4];
    char * condition = argv[5];
    int threadNum = stoi(argv[6]);
    threads = threadNum;
    if (argc == 8){
        pNo = stoi(argv[7]);
    }
    int c1 = 0;
    int c2 = 0;
    readFile(file1, file2, col1, col2, &c1, &c2, inputSize);
    auto start = std::chrono::high_resolution_clock::now();
    if (strcmp(funcName, "basicNestedLoop") == 0){
        basicLoop(col1, col2, c1, c2);
    } else if (strcmp(funcName, "blockedNestedLoop") == 0){
        loopOverBlock(col1, col2, c1, c2, cacheSize);
    } else if (strcmp(funcName, "partitionedSortSerial") == 0){
        //partitionedSortSerial(col1, col2, c1, c2, pNo);
    }
    else if (strcmp(funcName, "partitionedSortParallel") == 0){
        partitionedSortParallel(col1, col2, c1, c2, pNo);
    } else if(strcmp(funcName, "sortParallel") == 0){
        sortParallel(col1, col2, c1, c2, pNo);
    }
    else if (strcmp(funcName, "basicSort") == 0){
        basicSort(col1, col2, c1, c2);
    } else if (strcmp(funcName, "partitionedHashSerial") == 0) {
      //partitionedHash(col1, col2, c1, c2, pNo, true);  
    } else if (strcmp(funcName, "partitionedHashParallel") == 0) {
        partitionedHash(col1, col2, c1, c2, pNo, false);  
    }
    else if (strcmp(funcName, "basicHash") == 0) {
      //basicHash(col1, col2, c1, c2);  
    } else {
        cout << "Invalid function name" << endl;
    }
    //hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    std::cout << "Total Time: " << elapsed.count() << " ms" << std::endl;
    delete[] col1;
    delete[] col2;
    return 0;
}