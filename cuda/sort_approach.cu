#include "hip/hip_runtime.h"
#include<iostream>
#include<algorithm>
#include<omp.h>
#include<chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include "general.cuh"
//using namespace std;
using ::std::cout;
using ::std::endl;
void basicSort(int * col1, int * col2, int c1, int c2){
    int cnt = 0;
    std::sort(col1, col1 + c1);
    std::sort(col2, col2 + c2);
    int i = 0;
    int j = 0;
    for(; i < c1 && j < c2;){
        if (col1[i] < col2[j]){
            i += 1;
        }
        else if (col1[i] > col2[j]){
            j += 1;
        } else {
            i += 1;
            j += 1;
            cnt += 1;     
        }
    }
    cout << cnt << endl;
}

__global__ void partitionKernelSort(int * c1, int * c2,int n1, int n2, int partitions, int blocks, CustomContainer * containers1, CustomContainer * containers2){
    int blockSize1 = n1 / blocks;
    int p;
    int idx = blockSize1 * blockIdx.x + threadIdx.x;
    for (int i = idx; i < blockSize1 * (blockIdx.x + 1) && i < n1; i += blockDim.x){
        p = c1[i] % partitions;
        containers1->push_back(p, 0, c1[i]);
    }
    int blockSize2 = n2 / blocks;
    idx = blockSize2 * blockIdx.x + threadIdx.x;
    for (int i = idx; i < blockSize2 * (blockIdx.x + 1) && i < n2; i += blockDim.x){
        p = c2[i] % partitions;
        containers2->push_back(p, 0, c2[i]);
    }
}

void sortParallel(int * c1, int * c2, int n1, int n2, int partitions){
    
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    float milliseconds = 0;
    // hipEventRecord(start);
    
    thrust::device_vector<int> dCol1(c1, c1 + n1);
    thrust::device_vector<int> dCol2(c2, c2 + n2);
    thrust::sort(dCol1.begin(), dCol1.end());
    thrust::sort(dCol2.begin(), dCol2.end());
    
    thrust::device_vector<int> result(n1);
    // int * result = 0;
    // hipMalloc(&result, sizeof(int) * n1);
    thrust::device_vector<int>::iterator last = thrust::set_intersection(dCol1.begin(), dCol1.end(), dCol2.begin(), dCol2.end(), result.begin());
    
    //hipDeviceSynchronize();
    int match = thrust::distance(result.begin(), last);
    
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    
    // hipEventElapsedTime(&milliseconds, start, stop);
    
    cout << "Matched - " <<  match << endl;
}


void partitionedSortParallel(int * c1, int * c2, int n1, int n2, int partitions) { 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    thrust::device_vector<int> dCol1(c1, c1 + n1);
    thrust::device_vector<int> dCol2(c2, c2 + n2);
    int blocks = 80;
    int blockSize = n1/1;
    CustomContainer * containers1 = new CustomContainer(2 * n1/partitions, partitions, 1, NULL);
    CustomContainer * containers2 = new CustomContainer(2 * n2/partitions, partitions, 1, NULL);
    CustomContainer * deviceContainers1 = 0;
    CustomContainer * deviceContainers2 = 0;
    hipMalloc(&deviceContainers1, sizeof(CustomContainer));
    hipMalloc(&deviceContainers2, sizeof(CustomContainer));
    hipMemcpy(&deviceContainers1->container, &containers1->container, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers1->meta, &containers1->meta, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers1->head, &containers1->head, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers2->container, &containers2->container, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers2->meta, &containers2->meta, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&deviceContainers2->head, &containers2->head, sizeof(int *), hipMemcpyHostToDevice);
    int * finalCount = 0;
    hipMalloc(&finalCount, sizeof(int)*partitions);
    try{
        partitionKernelSort<<<blocks, 256>>>(thrust::raw_pointer_cast(dCol1.data()),thrust::raw_pointer_cast(dCol2.data()),n1,n2, partitions, blocks, deviceContainers1, deviceContainers2);
    } catch (thrust::system_error e){
        std::cout << "Error: " << e.what() << std::endl;
    }
    hipDeviceSynchronize();
    thrust::device_vector<int> result(n1);
    thrust::device_vector<int>::iterator first = result.begin();
    thrust::device_vector<int>::iterator last_temp = first;
    for(int p = 0; p < partitions; p++){
        int * col1 = containers1->getPartition(p);
        int * col2 = containers2->getPartition(p);
        int n1 = containers1->getPartitionSize(p);
        int n2 = containers2->getPartitionSize(p);
        
        thrust::device_vector<int> dCol1(col1, col1+n1);
        thrust::device_vector<int> dCol2(col2, col2 +n2);
        thrust::sort(dCol1.begin(), dCol1.end());
        thrust::sort(dCol2.begin(), dCol2.end());
        thrust::device_vector<int>::iterator last = thrust::set_intersection(dCol1.begin(), dCol1.end(), dCol2.begin(), dCol2.end(), last_temp);
        last_temp = last;
    }
    //hipDeviceSynchronize();
    hipEventRecord(stop);
    float milliseconds = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time elapsed: " << milliseconds << std::endl;
    cout << "Matched - " << thrust::distance(result.begin(), last_temp) << endl;
}